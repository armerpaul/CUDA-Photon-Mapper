#include "hip/hip_runtime.h"
/*
 * CPE 570 Final Project
 * Photon Mapper
 * Zoë Wood
 * Paul Armer (parmer)
 */

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "glm/glm.hpp"
#include <math.h>
#include <algorithm>
#include <assert.h>
//#include "Image.h"
#include "types.h"
#include "cudaPhotonMapper.h"
#include "kdtree.h"

// kdtree types from kd-tree code
// These are needed for kdtree functions to run on CUDA
struct kdnode {
	double *pos;
	int dir;
	void *data;

	struct kdnode *left, *right;
};

struct res_node {
	struct kdnode *item;
	double dist_sq;
	struct res_node *next;
};

struct kdtree {
	int dim;
	struct kdnode *root;
	struct kdhyperrect *rect;
	void (*destr)(void*);
};

struct kdres {
	struct kdtree *tree;
	struct res_node *rlist, *riter;
	int size;
};
 

Camera * camera, *cam_d;
RectLight *light, *l_d;
Plane * planes, *p_d;
Sphere * spheres, *s_d;
Photon * photonArray, *ph_d;
kdtree * kdTree;
struct kdres * kdresult;
int numPhotons, kdTreeIncomplete = 1;
float theta, stheta;

Camera* CameraInit();
PointLight* LightInit();
Sphere* CreateSpheres();
Plane* CreatePlanes();
__host__ __device__ Point CreatePoint(float x, float y, float z);
__host__ __device__ color_t CreateColor(float r, float g, float b);

__global__ void CUDAPhotonTrace(Plane * f, RectLight *l, Sphere * s, Photon * position);
__global__ void CUDARayTrace(Camera * cam, Plane * f, Sphere * s, kdtree * tree, uchar4 * pos);

__device__ Photon * kd_res_photonf(struct kdres *rset, float *pos);

__device__ color_t RayTrace(Ray r, Sphere* s, Plane* f, kdtree * tree);
__device__ Photon PhotonTrace(Photon p, Sphere* s, Plane* f);
__device__ color_t SphereShading(int sNdx, Ray r, Point p, Sphere* sphereList, PointLight* l);
__device__ color_t Shading(Ray r, Point p, Point normalVector, PointLight* l, color_t diffuse, color_t ambient, color_t specular); 
__device__ float SphereRayIntersection(Sphere* s, Ray r);
__device__ float PlaneRayIntersection(Plane* s, Ray r);

static void HandleError( hipError_t err, const char * file, int line)
{
	if(err !=hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

/* 
 *  Handles CUDA errors, taking from provided sample code on clupo site
 */
extern "C" void setup_scene()
{
	kdTree = kd_create(3);
	HANDLE_ERROR( hipDeviceSetCacheConfig(hipFuncCachePreferL1));
	camera = CameraInit();
	light = new RectLight();
	spheres = CreateSpheres();
	planes = CreatePlanes(); 
	numPhotons = light->width * PHOTON_DENSITY * light->height * PHOTON_DENSITY;
	photonArray = (Photon *) malloc(sizeof(Photon) * numPhotons * NUM_BOUNCES); 
	
	HANDLE_ERROR( hipMalloc((void**)&cam_d, sizeof(Camera)) );
	HANDLE_ERROR( hipMalloc(&p_d, sizeof(Plane)*NUM_PLANES) );
	HANDLE_ERROR( hipMalloc(&l_d, sizeof(RectLight)) );
	HANDLE_ERROR( hipMalloc(&s_d,  sizeof(Sphere)*NUM_SPHERES));
	HANDLE_ERROR( hipMalloc(&ph_d,  sizeof(Photon)*numPhotons*NUM_BOUNCES));

	HANDLE_ERROR( hipMemcpy(l_d, light, sizeof(RectLight), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(cam_d, camera,sizeof(Camera), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(p_d, planes,sizeof(Plane)*NUM_PLANES, hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );
	
	theta = 0;
	stheta = 0;
}

/*
 * Function Wrapper for the kernel that shoots out photons
 */
extern "C" void photonLaunch()
{
	Point move;

	//light->position.x -= 2 *sin(theta += .01);	

	//spheres[NUM_SPHERES-1].radius=5;
	//spheres[NUM_SPHERES-1].center=light->position;
	//spheres[NUM_SPHERES-1].ambient=CreateColor(1,0,0);
	//spheres[NUM_SPHERES-1].diffuse=CreateColor(1,1,1);
	//spheres[NUM_SPHERES-1].specular=CreateColor(1,1,1);

	HANDLE_ERROR( hipMemcpy(l_d, light, sizeof(RectLight), hipMemcpyHostToDevice) );

	HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );

	HANDLE_ERROR( hipMemcpy(ph_d, photonArray, sizeof(Photon)*numPhotons*NUM_BOUNCES, hipMemcpyHostToDevice) );

	// The Kernel Call
	dim3 gridSize((light->width * PHOTON_DENSITY + 15)/16, (light->height * PHOTON_DENSITY + 15)/16);
	dim3 blockSize(16,16);
	CUDAPhotonTrace<<< gridSize, blockSize  >>>(p_d, l_d, s_d, ph_d);
	hipDeviceSynchronize();

	 
	if (kdTreeIncomplete) {
		HANDLE_ERROR( hipMemcpy(photonArray, ph_d, sizeof(Photon)*numPhotons, hipMemcpyDeviceToHost) );

		kd_clear(kdTree);
		for(int i=0; i < numPhotons; i++) {
			assert(0 == kd_insert3(kdTree, photonArray[i].position.x, photonArray[i].position.y, photonArray[i].position.z, &photonArray[i]));
		
			//printf("%f %f %f to %f %f %f\n", photonArray[i].position.x, photonArray[i].position.y, photonArray[i].position.z,
			//		photonArray[i].direction.x, photonArray[i].direction.y, photonArray[i].direction.z);
		}
		kdTreeIncomplete = false;
	} else {
		// Might do something here...
	}
	
}

extern "C" void renderScene(uchar4 *pos)
{
	
	CUDARayTrace(cam_d, p_d, s_d, kdTree, pos);
	
	
}




/*
 * CUDA global function which performs ray tracing. Responsible for initializing and writing to output vector
 */
__global__ void CUDARayTrace(Camera * cam, Plane * f, Sphere * s, kdtree *tree, uchar4 * pos)
{
	float tanVal = tan(FOV/2);

	//CALCULATE ABSOLUTE ROW,COL
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	color_t returnColor;
	Ray r;

	//BOUNDARY CHECK
	if(row >= WINDOW_HEIGHT || col >= WINDOW_WIDTH)
		return;

	float rvaly = tanVal - (2 * tanVal / WINDOW_HEIGHT) * row;
	float rvalx = -1 * WINDOW_WIDTH / WINDOW_HEIGHT * tanVal + (2 * tanVal / WINDOW_HEIGHT) * col;
	//INIT RAY VALUES
	r.origin = cam->eye;
	r.direction = cam->lookAt;
	r.direction += (rvalx * cam->lookRight);
	r.direction += (rvaly * cam->lookUp);
	r.direction = glm::normalize(r.direction);
	//r.direction.y += tanVal - (2 * tanVal / WINDOW_HEIGHT) * row;
	//r.direction.x += -1 * WINDOW_WIDTH / WINDOW_HEIGHT * tanVal + (2 * tanVal / WINDOW_HEIGHT) * col;

	//RAY TRACE
	returnColor = RayTrace(r, s, f, tree);

	//CALC OUTPUT INDEX
	int index = row *WINDOW_WIDTH + col;

	//PLACE DATA IN INDEX
	pos[index].x = 0xFF * returnColor.r;
	pos[index].y = 0xFF * returnColor.g;
	pos[index].z = 0xFF * returnColor.b;
	pos[index].w = 0xFF * returnColor.f;

}

/*
 * Performs Ray tracing over all spheres for any ray r
 */
__device__ color_t RayTrace(Ray r, Sphere* s, Plane* f, kdtree * tree) {
	color_t color = CreateColor(0, 0, 0); 
	kdres * nearestPhotons;
	float t, smallest;
	int i = 0, closestSphere = -1, closestPlane = -1;
	//r.direction += r.origin; //Set back to normal
	Point normalVector;
	//FIND CLOSEST SPHERE ALONG RAY R
	while (i < NUM_SPHERES) {
		t = SphereRayIntersection(s + i, r);

		if (t > 0 && (closestSphere < 0 || t < smallest)) {
			smallest = t;
			closestSphere = i;
		}
		i++;
	}
	//r.direction -= r.origin;
	i=0;
	while (i < NUM_PLANES) {
		t = PlaneRayIntersection(f + i, r);
		if (t > 0 && ( (closestSphere < 0 && closestPlane < 0) || t < smallest)) {//POSSIBLE LOGIC FIX CLOSESTSPHERE >1
			smallest = t;
			closestSphere = -1;
			closestPlane = i;
		}
		i++;
	}

	//SETUP FOR SHADOW CALCULATIONS
	i = 0;
	Ray shadowRay;

	if(closestPlane > -1 || closestSphere > -1)
	{	
		float resPoint[3];
		float pos[] = {r.direction.x * smallest, 
				r.direction.y * smallest,
				 r.direction.z * smallest};
		nearestPhotons = kd_nearest_rangef(tree, pos, PHOTON_RANGE);
		float dist;
		Photon * data;
		while( !kd_res_end( nearestPhotons ) ) {
			data = kd_res_photonf( nearestPhotons, resPoint );
			dist = glm::distance(glm::vec3(pos[0], pos[1], pos[2]), glm::vec3(resPoint[0], resPoint[1], resPoint[2]));
			color.r += ((PHOTON_RANGE - dist) / PHOTON_RANGE) * data->color.r;
			color.g += ((PHOTON_RANGE - dist) / PHOTON_RANGE) * data->color.g;
			color.b += ((PHOTON_RANGE - dist) / PHOTON_RANGE) * data->color.b;
		}
	}
	
	return color;
}


/*
 * CUDA global function which performs photon mappning. Responsible for initializing and writing to output vector
 */
__global__ void CUDAPhotonTrace(Plane * f, RectLight * l, Sphere * s, Photon * pos)
{
	//CALCULATE ABSOLUTE ROW,COL
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	Photon ph = Photon();

	//BOUNDARY CHECK
	if(row >= l->height * PHOTON_DENSITY || col >= l->width * PHOTON_DENSITY)
		return;

	//INIT PHOTON VALUES - Doesn't support a moving light yet, makes assumptions about position
	ph.position = l->position;
	ph.position.x += (float)row / (float)(l->height);
	ph.position.z += (float)col / (float)(l->width);

	ph.direction.x = l->normal.x + (float)row / (float)l->height; //just in hopes that something interesting happens
	ph.direction.y = l->normal.y; // we'll just keep this one the same since rand ain't workin'
	ph.direction.z = l->normal.z + (float)col / (float)l->width;
	ph.direction = glm::normalize(ph.direction);

	//CALC OUTPUT INDEX
	int index = row * l->height * PHOTON_DENSITY + col;

	//PLACE PHOTON IN INDEX
	pos[index] = PhotonTrace(ph, s, f);

}



/*
 * Performs Ray tracing over all spheres for any ray r
 */
__device__ Photon PhotonTrace(Photon ph, Sphere* s, Plane* f) {
	
	color_t color = CreateColor(0, 0, 0); 
	float t, smallest;
	int i = 0, closestSphere = -1, closestPlane = -1;
	//r.direction += r.origin; //Set back to normal
	Point normalVector;
	Ray r;
	r.origin = ph.position;
	r.direction = ph.direction;
	//FIND CLOSEST SPHERE ALONG RAY R
	while (i < NUM_SPHERES) {
		t = SphereRayIntersection(s + i, r);

		if (t > 0 && (closestSphere < 0 || t < smallest)) {
			smallest = t;
			closestSphere = i;
		}
		i++;
	}
	//r.direction -= r.origin;
	i=0;
	while (i < NUM_PLANES) {
		t = PlaneRayIntersection(f + i, r);
		if (t > 0 && ( (closestSphere < 0 && closestPlane < 0) || t < smallest)) {//POSSIBLE LOGIC FIX CLOSESTSPHERE >1
			smallest = t;
			closestSphere = -1;
			closestPlane = i;
		}
		i++;
	}

	if (closestSphere > -1) {
		ph.position.x += smallest * ph.direction.x;
		ph.position.y += smallest * ph.direction.y;
		ph.position.z += smallest * ph.direction.z;

		ph.color = s[closestSphere].ambient;
	} else if (closestPlane > -1) {
		ph.position.x += smallest * ph.direction.x;
		ph.position.y += smallest * ph.direction.y;
		ph.position.z += smallest * ph.direction.z;	
		ph.color = f[closestPlane].ambient;
	}

	return ph;

}

/*
 * Determines distance of intersection of Ray with Plane, -1 returned if no intersection
 */
__device__ float PlaneRayIntersection(Plane *p, Ray r)
{
	float t;
	//Point N = glm::normalize(p->normal);
	float denominator = glm::dot(r.direction,p->normal);
	if(denominator!=0)
	{
		t = (glm::dot(p->center-r.origin,p->normal)) / denominator;
		if (t>1000000)
			return -1;
		return t;
	}
	else
	{
		return -1;
	}
}


/*
 * Determines distance of intersection of Ray with Sphere, -1 returned if no intersection
 * http://sci.tuomastonteri.fi/programming/sse/example3
 */
__device__ float SphereRayIntersection(Sphere* s, Ray r) {
	float a, b, c, d, t1, t2;

	a = glm::dot((r.direction), (r.direction));

	b = glm::dot((r.origin)-(s->center),(r.direction));
	c = glm::dot((s->center),(s->center)) +glm::dot(r.origin,r.origin) -2.0f*glm::dot(r.origin, s->center)
		- (s->radius * s->radius);
	d = (b * b) - (a * c);

	if (d >= 0) {

		t1 = (-1 * b - sqrt(d)) / (a);
		t2 = (-1 * b + sqrt(d)) / (a);

		if (t2 > t1 && t1 > 0) {
			return t1;

		} else if (t2 > 0) {
			return t2;
		}
	}
	return -1;
}


extern "C" void ijklMove(unsigned char key)
{
	float sin_theta_x, cos_theta_x, sin_theta_y,cos_theta_y;
	switch(key){
	case('i'):
		camera->theta_x+=.05;
		break; 
	case('k'):
		camera->theta_x-=.05;
		break;
	case('j'):
		camera->theta_y-=.05;
		break;
	case('l'):
		camera->theta_y+=.05;
		break;
	}
	sin_theta_x = sin(camera->theta_x);
	sin_theta_y = sin(camera->theta_y);
	cos_theta_x = cos(camera->theta_x);
	cos_theta_y = cos(camera->theta_y);

	camera->lookAt = glm::normalize(CreatePoint(sin_theta_y ,sin_theta_x , -1*cos_theta_x*cos_theta_y));
	camera->lookRight = glm::normalize(CreatePoint(cos_theta_y , 0 , sin_theta_y));
	camera->lookUp = glm::normalize(CreatePoint(0,cos_theta_x, sin_theta_x));
}



extern "C" void wasdMove(unsigned char key)
{
	Point move;
	switch(key){
	case('w'):
		move = 10.f * camera->lookAt;
		break; 
	case('s'):
		move = -10.f *camera->lookAt;
		break;
	case('a'):
		move = -10.f * camera->lookRight;
		break;
	case('d'):
		move = 10.f * camera->lookRight;
		break;
	}
	camera->eye += move;
}
extern "C" void misc(unsigned char key)
{
	Point center;
	switch(key){
	case('q'):
		{
			// just for testing - resets kdTree
			kdTreeIncomplete = 1;
			camera = CameraInit();
			break;
		}
	case('r'):
		{
			spheres = CreateSpheres();
			HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );
			break;
		}
	case('-'):
		{
			for(int i = 0; i < NUM_SPHERES; i++)
				spheres[i].radius = glm::max(0.f, spheres[i].radius-1);
			HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );
			break;
		}
	case('='):
		{
			for(int i = 0; i < NUM_SPHERES; i++)
				spheres[i].radius = glm::min(100.f, spheres[i].radius+1);
			HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );
			break;
		}
	case('o'):
		{
			Point center = *new Point(0,0,-2400);
			center = *new Point(0,0,-2400);
			for(int i = 0; i < NUM_SPHERES; i++)
			{
				Point c_dir = glm::normalize(spheres[i].center - center);
				Point move_dir = glm::cross(c_dir, *new Point(0,1,0));
				spheres[i].center += 5.f*move_dir;
				spheres[i].center -= 5.f*c_dir;

			}
			HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );
			break;
		}
	case('p'):
		{
			center = *new Point(0,0,-2400);
			for(int i = 0; i < NUM_SPHERES; i++)
			{
				Point c_dir = glm::normalize(spheres[i].center - center);
				Point move_dir = glm::cross(c_dir, *new Point(0,1,0));
				spheres[i].center -= 10.f*move_dir;
				spheres[i].center += 10.f*c_dir;

			}
			HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );
			break;
		}
	case('['):
		{
			center = camera->eye;
			for(int i = 0; i < NUM_SPHERES; i++)
			{
				Point c_dir = glm::normalize(spheres[i].center - center);
				Point move_dir = glm::cross(c_dir, *new Point(0,1,0));
				spheres[i].center += 10.f*move_dir;
				//spheres[i].center -= 10.f*c_dir;

			}
			HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );
			break;
		}

	case(']'):
		{
			center = camera->eye;
			for(int i = 0; i < NUM_SPHERES; i++)
			{
				Point c_dir = glm::normalize(spheres[i].center - center);
				Point move_dir = glm::cross(c_dir, *new Point(0,1,0));
				spheres[i].center -= 10.f*move_dir;
				//spheres[i].center += 10.f*c_dir;

			}
			HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );
			break;
		}
	case('9'):
		{

			for(int i = 0; i < NUM_PLANES; i++) {
				planes[i].ambient.r = glm::max(planes[i].ambient.r - .05f, 0.f);
				planes[i].ambient.g = glm::max(planes[i].ambient.g - .05f, 0.f);
				planes[i].ambient.b = glm::max(planes[i].ambient.b - .05f, 0.f);
				planes[i].diffuse.r = glm::max(planes[i].diffuse.r - .05f, 0.f);
				planes[i].diffuse.g = glm::max(planes[i].diffuse.g - .05f, 0.f);
				planes[i].diffuse.b = glm::max(planes[i].diffuse.b - .05f, 0.f);
			}

			HANDLE_ERROR( hipMemcpy(p_d, planes,sizeof(Plane)*NUM_PLANES, hipMemcpyHostToDevice) );
			break;
		}
	case('0'):
		{
			for(int i = 0; i < NUM_PLANES; i++) {
				planes[i].ambient.r = glm::min(planes[i].ambient.r + .05f, 1.f);
				planes[i].ambient.g = glm::min(planes[i].ambient.g + .05f, 1.f);
				planes[i].ambient.b = glm::min(planes[i].ambient.b + .05f, 1.f);
				planes[i].diffuse.r = glm::min(planes[i].diffuse.r + .05f, 1.f);
				planes[i].diffuse.g = glm::min(planes[i].diffuse.g + .05f, 1.f);
				planes[i].diffuse.b = glm::min(planes[i].diffuse.b + .05f, 1.f);
			}

			HANDLE_ERROR( hipMemcpy(p_d, planes,sizeof(Plane)*NUM_PLANES, hipMemcpyHostToDevice) );
			break;
		}
	}
}


__device__ float fastSqrt( float number )
{
        long i;
        float x2, y;
        const float threehalfs = 1.5F;
 
        x2 = number * 0.5F;
        y  = number;
        i  = * ( long * ) &y;                       // evil floating point bit level hacking
        i  = 0x5f3759df - ( i >> 1 );               // what the actual fuck?
        y  = * ( float * ) &i;
        y  = y * ( threehalfs - ( x2 * y * y ) );   // 1st iteration
//      y  = y * ( threehalfs - ( x2 * y * y ) );   // 2nd iteration, this can be removed
 
        return 1 / y;
}
