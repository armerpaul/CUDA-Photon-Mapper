#include "hip/hip_runtime.h"
/*
 * CPE 570 && CPE 458 Duet
 * Ray Tracer
 * Professor Christopher Lupo and Professor Zoe" Wood
 * Paul Armer(parmer), Bryan Ching(bcching), Matt Crussell(macrusse)
 */

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <algorithm>
#include "glm/glm.hpp"
#include "types.h"
#include "kdtree.h"
#include "cudaRayTrace.h"

Camera * camera, *cam_d;
Plane * planes, *p_d;
Sphere * spheres, *s_d;
RectLight * rectlight, *rl_d;
Photon * photonArray, * ph_d;
Point * points, *pt_d;
kdtree * tree;
int numPhotons;
float theta, stheta;

Camera* CameraInit();
PointLight* LightInit();
Sphere* CreateSpheres();
Plane* CreatePlanes();
RectLight* RectLightInit();
__host__ __device__ Point CreatePoint(float x, float y, float z);
__host__ __device__ color_t CreateColor(float r, float g, float b);

__global__ void CUDARayTrace(Camera * cam, Plane * f, Sphere * s, Point * pts, int offsetX, int offsetY);
__global__ void CUDAPhotonTrace(Plane * f, RectLight * l, Sphere * s, Photon * photons);

__device__ Photon PhotonTrace(Photon ph, Sphere * s, Plane * f, RectLight * rl);
__device__ Point RayTrace(Ray r, Sphere* s, Plane* f);
__device__ color_t SphereShading(int sNdx, Ray r, Point p, Sphere* sphereList, PointLight* l);
__device__ color_t Shading(Ray r, Point p, Point normalVector, PointLight* l, color_t diffuse, color_t ambient, color_t specular); 
__device__ float SphereRayIntersection(Sphere* s, Ray r);
__device__ float PlaneRayIntersection(Plane* s, Ray r);

static void HandleError( hipError_t err, const char * file, int line)
{
	if(err !=hipSuccess){
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

/* 
 *  Handles CUDA errors, taking from provided sample code on clupo site
 */
extern "C" void setup_scene()
{
	HANDLE_ERROR( hipDeviceSetCacheConfig(hipFuncCachePreferL1));
	camera = CameraInit();
	rectlight = RectLightInit();
	spheres = CreateSpheres();
	planes = CreatePlanes(); 
	rectlight = RectLightInit();
	
	numPhotons = PHOTON_DENSITY * PHOTON_DENSITY * rectlight->width * rectlight->height;
	photonArray = new Photon[numPhotons];
	points = new Point[1024]; // 32 * 32
	tree = kd_create(3);
	
	HANDLE_ERROR( hipMalloc(&cam_d, sizeof(Camera)) );
	HANDLE_ERROR( hipMalloc(&p_d, sizeof(Plane)*NUM_PLANES) );
	HANDLE_ERROR( hipMalloc(&s_d,  sizeof(Sphere)*NUM_SPHERES));
	HANDLE_ERROR( hipMalloc(&rl_d,  sizeof(RectLight)));
	HANDLE_ERROR( hipMalloc(&ph_d, sizeof(Photon) * numPhotons) );
	HANDLE_ERROR( hipMalloc(&pt_d, sizeof(Point) * 32 * 32) );
	
	HANDLE_ERROR( hipMemcpy(cam_d, camera,sizeof(Camera), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(p_d, planes,sizeof(Plane)*NUM_PLANES, hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(rl_d, rectlight,sizeof(RectLight), hipMemcpyHostToDevice) );
	
	theta = 0;
	stheta = 0;
}


extern "C" void launch_kernel(uchar4* pos, unsigned int image_width, 
		unsigned int image_height, float time)
{
	Point move;

	//light->position.x -= 2 *sin(theta += .01);	
	HANDLE_ERROR( hipMemcpy(rl_d, rectlight,sizeof(RectLight), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(cam_d, camera,sizeof(Camera), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );

	// == FIRST PASS == //
	// Fire le protons! 
	dim3 gridSize2((PHOTON_DENSITY * rectlight->width + 15)/16, (PHOTON_DENSITY * rectlight->width +15)/16);
	dim3 blockSize2(16,16);
	CUDAPhotonTrace<<< gridSize2, blockSize2 >>>(p_d, rl_d, s_d, ph_d);
	hipDeviceSynchronize();
	
	// Make a kd-tree from the array of photons
	kd_clear(tree);
	HANDLE_ERROR( hipMemcpy(photonArray, ph_d, numPhotons * sizeof(Photon), hipMemcpyDeviceToHost) );
	for (int i = 0; i < numPhotons; i++) {
		kd_insert3f(tree, photonArray[i].ray.origin.x, photonArray[i].ray.origin.y, photonArray[i].ray.origin.z, &photonArray[i]);
	}
	
	struct kdres *presults;
	Photon * data;
	float point[3], dist;
	dim3 gridSize(2, 2); //since we have to transfer the points back and forth we need to split it up.
	dim3 blockSize(16,16);
	
	printf("second pass\n");
	// == SECOND PASS == //
	for (int i = 0; i < (WINDOW_WIDTH + 31)/32; i++) {
		for (int j = 0; j < (WINDOW_HEIGHT + 31)/32; j++) {
			// Find the view ray intersection points
			CUDARayTrace<<< gridSize, blockSize >>>(cam_d, p_d, s_d, pt_d, i, j);
			hipDeviceSynchronize();
			
			HANDLE_ERROR( hipMemcpy(points, pt_d, 1024 * sizeof(Point), hipMemcpyDeviceToHost) );
			for (long i = 0; i < WINDOW_WIDTH * WINDOW_HEIGHT; i++) {
				presults = kd_nearest_range3f( tree, photonArray[i].ray.origin.x, 
													 photonArray[i].ray.origin.y,
													 photonArray[i].ray.origin.z, 1.f);

				printf( "found %d results:\n", kd_res_size(presults) );
				int limit = 0;
				while( !kd_res_end( presults ) && limit++ < 10) {
					/* get the data and position of the current result item */
					data = (Photon*)kd_res_itemf( presults, point );
					
					/* compute the distance of the current result from the pt */
					dist = glm::distance(glm::vec3(points[i].x, points[i].y, points[i].z), 
										 glm::vec3(point[0], point[1], point[2] ));

					/* print out the retrieved data */
					printf("distance: %f", dist);

					/* go to the next entry */
					kd_res_next( presults );
				}
				kd_res_free(presults);
			}
			
		}
	}

	
	
	
	// Calculate color of each point
	
	
	// I have to do the color assignment on the GPU because texture is created there -- WILL FIX *crosses fingers*
	
} 


/*
 * CUDA global function that shoots out the photons!
 */
__global__ void CUDAPhotonTrace(Plane * f, RectLight * l, Sphere * s, Photon * photons)
{
	float tanVal = tan(PHOTON_SPREAD/2);

	//CALCULATE ABSOLUTE ROW,COL
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	Photon ph;

	//BOUNDARY CHECK
	if(row >= l->height || col >= l->width)
		return;

	float rvaly = tanVal - (2 * tanVal / l->height) * row;
	float rvalx = -1 * l->width / l->height * tanVal + (2 * tanVal / l->height) * col;
	//INIT RAY VALUES
	ph.ray.origin = l->position;
	ph.ray.direction = l->position + l->normal;
	ph.ray.direction += (rvalx * l->wVec);
	ph.ray.direction += (rvaly * l->hVec);
	ph.ray.direction = glm::normalize(ph.ray.direction);

	//CALC OUTPUT INDEX
	int index = row * l->width + col;

	//RAY TRACE
	photons[index] = PhotonTrace(ph, s, f, l);
}

__device__ Photon PhotonTrace(Photon ph, Sphere * s, Plane * f, RectLight * rl) 
{
	Photon tempPh;
	float t, smallest = 0.f;
	int i = 0, closestSphere = -1, closestPlane = -1;
	Point normalVector;
	//FIND CLOSEST SPHERE ALONG RAY R
	while (i < NUM_SPHERES) {
		t = SphereRayIntersection(s + i, ph.ray);

		if (t > 0 && (closestSphere < 0 || t < smallest)) {
			smallest = t;
			closestSphere = i;
		}
		i++;
	}

	i=0;
	while (i < NUM_PLANES) {
		t = PlaneRayIntersection(f + i, ph.ray);
		if (t > 0 && ( (closestSphere < 0 && closestPlane < 0) || t < smallest)) {//POSSIBLE LOGIC FIX CLOSESTSPHERE >1
			smallest = t;
			closestSphere = -1;
			closestPlane = i;
		}
		i++;
	}

	//SETUP FOR SHADOW CALCULATIONS
	i = 0;
	
	tempPh.ray.origin = CreatePoint(ph.ray.direction.x * smallest, ph.ray.direction.y * smallest, ph.ray.direction.z * smallest);
	tempPh.ray.direction = rl->normal;

	//DETERMINE IF SPHERE IS BLOCKING RAY FROM LIGHT TO SPHERE
	if(closestSphere > -1 )
	{
		tempPh.color.r = s[closestSphere].diffuse.r + s[closestSphere].ambient.r;
		tempPh.color.g = s[closestSphere].diffuse.g + s[closestSphere].ambient.g;
		tempPh.color.b = s[closestSphere].diffuse.b + s[closestSphere].ambient.b;
	}
	else if (closestPlane > -1) {
		tempPh.color.r = s[closestSphere].diffuse.r + s[closestSphere].ambient.r;
		tempPh.color.g = s[closestSphere].diffuse.g + s[closestSphere].ambient.g;
		tempPh.color.b = s[closestSphere].diffuse.b + s[closestSphere].ambient.b;
	}
	else {
		tempPh.color = CreateColor(0, 0, 0);
	}


	return tempPh;
}

/*
 * CUDA global function which performs ray tracing. Responsible for initializing and writing to output vector
 */
__global__ void CUDARayTrace(Camera * cam, Plane * f, Sphere * s, Point * pts, int offsetX, int offsetY)
{
	float tanVal = tan(FOV/2);

	//CALCULATE ABSOLUTE ROW,COL
	int row = blockIdx.y * blockDim.y + threadIdx.y + offsetY * 32;
	int col = blockIdx.x * blockDim.x + threadIdx.x + offsetX * 32;
	Ray r;

	//BOUNDARY CHECK
	if(row >= WINDOW_HEIGHT || col >= WINDOW_WIDTH)
		return;

	float rvaly = tanVal - (2 * tanVal / WINDOW_HEIGHT) * row;
	float rvalx = -1 * WINDOW_WIDTH / WINDOW_HEIGHT * tanVal + (2 * tanVal / WINDOW_HEIGHT) * col;
	//INIT RAY VALUES
	r.origin = cam->eye;
	r.direction = cam->lookAt;
	r.direction += (rvalx * cam->lookRight);
	r.direction += (rvaly * cam->lookUp);
	r.direction = glm::normalize(r.direction);
	//r.direction.y += tanVal - (2 * tanVal / WINDOW_HEIGHT) * row;
	//r.direction.x += -1 * WINDOW_WIDTH / WINDOW_HEIGHT * tanVal + (2 * tanVal / WINDOW_HEIGHT) * col;

	//CALC OUTPUT INDEX
	pts[threadIdx.x * 32 * threadIdx.y] = RayTrace(r, s, f);
}

/*
 * Performs Ray tracing over all spheres for any ray r
 */
__device__ Point RayTrace(Ray r, Sphere* s, Plane* f) {
	Point pt; 
	float t, smallest = 0;
	int i = 0, closestSphere = -1, closestPlane = -1;
	//r.direction += r.origin; //Set back to normal
	Point normalVector;
	//FIND CLOSEST SPHERE ALONG RAY R
	while (i < NUM_SPHERES) {
		t = SphereRayIntersection(s + i, r);

		if (t > 0 && (closestSphere < 0 || t < smallest)) {
			smallest = t;
			closestSphere = i;
		}
		i++;
	}
	//r.direction -= r.origin;
	i=0;
	while (i < NUM_PLANES) {
		t = PlaneRayIntersection(f + i, r);
		if (t > 0 && ( (closestSphere < 0 && closestPlane < 0) || t < smallest)) {//POSSIBLE LOGIC FIX CLOSESTSPHERE >1
			smallest = t;
			closestSphere = -1;
			closestPlane = i;
		}
		i++;
	}

	//SETUP FOR SHADOW CALCULATIONS
	i = 0;
	return CreatePoint(r.direction.x * smallest, r.direction.y * smallest, r.direction.z * smallest);
}

/*
 * Determines distance of intersection of Ray with Plane, -1 returned if no intersection
 */
__device__ float PlaneRayIntersection(Plane *p, Ray r)
{
	float t;
	//Point N = glm::normalize(p->normal);
	float denominator = glm::dot(r.direction,p->normal);
	if(denominator!=0)
	{
		t = (glm::dot(p->center-r.origin,p->normal)) / denominator;
		if (t>1000000)
			return -1;
		return t;
	}
	else
	{
		return -1;
	}
}


/*
 * Determines distance of intersection of Ray with Sphere, -1 returned if no intersection
 * http://sci.tuomastonteri.fi/programming/sse/example3
 */
__device__ float SphereRayIntersection(Sphere* s, Ray r) {
	float a, b, c, d, t1, t2;

	a = glm::dot((r.direction), (r.direction));

	b = glm::dot((r.origin)-(s->center),(r.direction));
	c = glm::dot((s->center),(s->center)) +glm::dot(r.origin,r.origin) -2.0f*glm::dot(r.origin, s->center)
		- (s->radius * s->radius);
	d = (b * b) - (a * c);

	if (d >= 0) {

		t1 = (-1 * b - sqrt(d)) / (a);
		t2 = (-1 * b + sqrt(d)) / (a);

		if (t2 > t1 && t1 > 0) {
			return t1;

		} else if (t2 > 0) {
			return t2;

		}
	}
	return -1;
}
/*
 * Determines Ambient, Diffuse, and Specular lighting on the plane
 */ 
__device__ color_t Shading(Ray r, Point p, Point normalVector,
		PointLight* l, color_t diffuse, color_t ambient, color_t specular) {
	color_t a, d, s, total;
	float NdotL, RdotV;
	Point viewVector, lightVector, reflectVector;

	viewVector = glm::normalize((r.origin)-p);

	lightVector = glm::normalize((l->position) -p);

	NdotL = glm::dot(lightVector, normalVector);
	reflectVector = (2.f *normalVector*NdotL) -lightVector;

	a.r = l->ambient.r * ambient.r;
	a.g = l->ambient.g * ambient.g;
	a.b = l->ambient.b * ambient.b;

	// Diffuse
	d.r = NdotL * l->diffuse.r * diffuse.r * (NdotL > 0);
	d.g = NdotL * l->diffuse.g * diffuse.g * (NdotL > 0);
	d.b = NdotL * l->diffuse.b * diffuse.b * (NdotL > 0);

	// Specular
	RdotV = glm::pow(glm::dot(glm::normalize(reflectVector), viewVector), 100.f);
	s.r = RdotV * l->specular.r * specular.r * (NdotL > 0) *(RdotV>0);
	s.g = RdotV * l->specular.g * specular.g * (NdotL > 0) *(RdotV>0);
	s.b = RdotV * l->specular.b * specular.b * (NdotL > 0) *(RdotV>0);

	total.r = glm::min(1.f, a.r + d.r + s.r);
	total.g = glm::min(1.f, a.g + d.g + s.g);
	total.b = glm::min(1.f, a.b + d.b + s.b);
	total.f = 1.f;
	return total;
}

/*
 * Initializes camera at point (X,Y,Z)
 */
Camera* CameraInit() {

	Camera* c = new Camera();

	c->eye = CreatePoint(0, 0, 0);//(X,Y,Z)
	c->lookAt = CreatePoint(0, 0, SCREEN_DISTANCE);
	c->lookUp = CreatePoint(0, 1, 0);
	c->lookRight = CreatePoint(1, 0, 0);
	c->theta_x = 0;
	c->theta_y = 0;
	return c;
}

/*
 * Initializes light at hardcoded (X,Y,Z) coordinates
 */
PointLight* LightInit() {
	PointLight* l = new PointLight();

	l->ambient = CreateColor(0.2, 0.2, 0.2);
	l->diffuse = CreateColor(0.6, 0.6, 0.6);
	l->specular = CreateColor(0.4, 0.4, 0.4);

	l->position = CreatePoint(50, 50, -400);

	return l;
}

/*
 * Initializes the rectangular photon emitting light. pew pew
 */
RectLight * RectLightInit() {
	RectLight * rl = new RectLight();
	
	rl->position = CreatePoint(0, 100, -100);
	rl->color = CreateColor(1.f, 1.f, 1.f);
	rl->width = 10.f;
	rl->height = 10.f;
	rl->normal = CreatePoint(0, -1, 0);
	
	return rl;
}

/*
 * Creates a point, for GLM Point has been defined as vec3
 */
__host__  __device__ Point CreatePoint(float x, float y, float z) {
	Point p;

	p.x = x;
	p.y = y;
	p.z = z;

	return p;
}

/*
 * Creates a color_t type color based on input values
 */
__host__ __device__ color_t CreateColor(float r, float g, float b) {
	color_t c;

	c.r = r;
	c.g = g;
	c.b = b;
	c.f = 1.0;

	return c;
}

/*
 * Creates NUM_SPHERES # of Spheres, with randomly chosen values on color, location, and size
 */
Sphere* CreateSpheres() {
	Sphere* spheres = new Sphere[NUM_SPHERES]();
	float randr, randg, randb;
	int num = 0;
	while (num < NUM_SPHERES) {
		randr = (rand()%1000) /1000.f ;
		randg = (rand()%1000) /1000.f ;
		randb = (rand()%1000) /1000.f ;
		spheres[num].radius = 80. - rand() % 60;
		spheres[num].center = CreatePoint(2400. - rand() % 4800,
				700 - rand() % 1100,
				-0. - rand() %4800);
		spheres[num].ambient = CreateColor(randr, randg, randb);
		spheres[num].diffuse = CreateColor(randr, randg, randb);
		spheres[num].specular = CreateColor(1., 1., 1.);
		num++;
	}

	return spheres;
}

/*
 * Creates NUM_PLANES NUMBER OF PLANES, CURRENTLY THIS IS HARDCODED
 */
Plane* CreatePlanes() {
	Plane* planes = new Plane[NUM_PLANES]();
	planes[0].normal = CreatePoint(0,1,0);
	planes[0].center = CreatePoint(0,-500,0);
	planes[0].ambient = planes[0].diffuse = planes[0].specular = CreateColor(1,1,1);

	planes[1].normal = CreatePoint(0,-1,0) ;
	planes[1].center = CreatePoint(0,800,0);
	planes[1].ambient = planes[1].diffuse = planes[1].specular = CreateColor(1,1,1);

	planes[2].normal = CreatePoint(0,0, 1) ;
	planes[2].center = CreatePoint(0,0,-5000);
	planes[2].ambient = planes[2].diffuse = planes[2].specular = CreateColor(1,1,1);

	planes[3].normal = CreatePoint(1,0,0) ;
	planes[3].center = CreatePoint(-2400,0,0);
	planes[3].ambient = planes[3].diffuse = planes[3].specular = CreateColor(1,1,1);

	planes[4].normal = CreatePoint(-1,0,0) ;
	planes[4].center = CreatePoint(2400,0, 0);
	planes[4].ambient = planes[4].diffuse = planes[4].specular = CreateColor(1,1,1);

	planes[5].normal = CreatePoint(0,0,-1) ;
	planes[5].center = CreatePoint(0,0, 1000);
	planes[5].ambient = planes[5].diffuse = planes[5].specular = CreateColor(1,1,1);

	return planes;
}


extern "C" void ijklMove(unsigned char key)
{
	float sin_theta_x, cos_theta_x, sin_theta_y,cos_theta_y;
	switch(key){
	case('i'):
		camera->theta_x+=.05;
		break; 
	case('k'):
		camera->theta_x-=.05;
		break;
	case('j'):
		camera->theta_y-=.05;
		break;
	case('l'):
		camera->theta_y+=.05;
		break;
	}
	sin_theta_x = sin(camera->theta_x);
	sin_theta_y = sin(camera->theta_y);
	cos_theta_x = cos(camera->theta_x);
	cos_theta_y = cos(camera->theta_y);

	camera->lookAt = glm::normalize(CreatePoint(sin_theta_y ,sin_theta_x , -1*cos_theta_x*cos_theta_y));
	camera->lookRight = glm::normalize(CreatePoint(cos_theta_y , 0 , sin_theta_y));
	camera->lookUp = glm::normalize(CreatePoint(0,cos_theta_x, sin_theta_x));
}



extern "C" void wasdMove(unsigned char key)
{
	Point move;
	switch(key){
	case('w'):
		move = 10.f * camera->lookAt;
		break; 
	case('s'):
		move = -10.f *camera->lookAt;
		break;
	case('a'):
		move = -10.f * camera->lookRight;
		break;
	case('d'):
		move = 10.f * camera->lookRight;
		break;
	}
	camera->eye += move;
}
extern "C" void misc(unsigned char key)
{
	Point center;
	switch(key){
	case('q'):
		{
			camera = CameraInit();
			break;
		}
	case('r'):
		{
			spheres = CreateSpheres();
			HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );
			break;
		}
	case('-'):
		{
			for(int i = 0; i < NUM_SPHERES; i++)
				spheres[i].radius = glm::max(0.f, spheres[i].radius-1);
			HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );
			break;
		}
	case('='):
		{
			for(int i = 0; i < NUM_SPHERES; i++)
				spheres[i].radius = glm::min(100.f, spheres[i].radius+1);
			HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );
			break;
		}
	case('o'):
		{
			Point center = *new Point(0,0,-2400);
			center = *new Point(0,0,-2400);
			for(int i = 0; i < NUM_SPHERES; i++)
			{
				Point c_dir = glm::normalize(spheres[i].center - center);
				Point move_dir = glm::cross(c_dir, *new Point(0,1,0));
				spheres[i].center += 5.f*move_dir;
				spheres[i].center -= 5.f*c_dir;

			}
			HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );
			break;
		}
	case('p'):
		{
			center = *new Point(0,0,-2400);
			for(int i = 0; i < NUM_SPHERES; i++)
			{
				Point c_dir = glm::normalize(spheres[i].center - center);
				Point move_dir = glm::cross(c_dir, *new Point(0,1,0));
				spheres[i].center -= 10.f*move_dir;
				spheres[i].center += 10.f*c_dir;

			}
			HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );
			break;
		}
	case('['):
		{
			center = camera->eye;
			for(int i = 0; i < NUM_SPHERES; i++)
			{
				Point c_dir = glm::normalize(spheres[i].center - center);
				Point move_dir = glm::cross(c_dir, *new Point(0,1,0));
				spheres[i].center += 10.f*move_dir;
				//spheres[i].center -= 10.f*c_dir;

			}
			HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );
			break;
		}

	case(']'):
		{
			center = camera->eye;
			for(int i = 0; i < NUM_SPHERES; i++)
			{
				Point c_dir = glm::normalize(spheres[i].center - center);
				Point move_dir = glm::cross(c_dir, *new Point(0,1,0));
				spheres[i].center -= 10.f*move_dir;
				//spheres[i].center += 10.f*c_dir;

			}
			HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );
			break;
		}
	case('9'):
		{

			for(int i = 0; i < NUM_PLANES; i++) {
				planes[i].ambient.r = glm::max(planes[i].ambient.r - .05f, 0.f);
				planes[i].ambient.g = glm::max(planes[i].ambient.g - .05f, 0.f);
				planes[i].ambient.b = glm::max(planes[i].ambient.b - .05f, 0.f);
				planes[i].diffuse.r = glm::max(planes[i].diffuse.r - .05f, 0.f);
				planes[i].diffuse.g = glm::max(planes[i].diffuse.g - .05f, 0.f);
				planes[i].diffuse.b = glm::max(planes[i].diffuse.b - .05f, 0.f);
			}

			HANDLE_ERROR( hipMemcpy(p_d, planes,sizeof(Plane)*NUM_PLANES, hipMemcpyHostToDevice) );
			break;
		}
	case('0'):
		{
			for(int i = 0; i < NUM_PLANES; i++) {
				planes[i].ambient.r = glm::min(planes[i].ambient.r + .05f, 1.f);
				planes[i].ambient.g = glm::min(planes[i].ambient.g + .05f, 1.f);
				planes[i].ambient.b = glm::min(planes[i].ambient.b + .05f, 1.f);
				planes[i].diffuse.r = glm::min(planes[i].diffuse.r + .05f, 1.f);
				planes[i].diffuse.g = glm::min(planes[i].diffuse.g + .05f, 1.f);
				planes[i].diffuse.b = glm::min(planes[i].diffuse.b + .05f, 1.f);
			}

			HANDLE_ERROR( hipMemcpy(p_d, planes,sizeof(Plane)*NUM_PLANES, hipMemcpyHostToDevice) );
			break;
		}
	}
}
